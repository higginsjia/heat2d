#include "hip/hip_runtime.h"
#include <stdio.h>
#include <lcutil.h>
#include <timestamp.h>
#include <string.h>
#include "lib.h"

__global__ void heat(float* A, float* B, int matrixSize, int matrixSize2) {
	
	/*Compute row and column in the matrix*/
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	/*Translate the 2d indexes in 1d index*/
	int i = row + col * matrixSize;
	
	/*Make the operations if needed*/
	if ((row != 0) && (row != matrixSize - 1) &&
	(col != 0) && (col != matrixSize -1) &&
	i < matrixSize2) {
		B[i] = A[i] +
		0.1 * (A[i - 1] + A[i + 1] - 2 * A[i]) +
		0.1 * (A[i - matrixSize] + A[i + matrixSize] - 2 * A[i]);
	}
}

extern "C" float heat2DGPU(float** A, float** B, int matrixSize, int steps, int threads) {
	
	int i;
	
	/*Cuda matrices creation and initialization*/
	float *Aa, *Bb;
	CUDA_SAFE_CALL( hipMalloc((void**)&Aa, matrixSize*matrixSize*sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void**)&Bb, matrixSize*matrixSize*sizeof(float)) );
	
	
	/*Copy data to device memory*/
	for (i = 0; i < matrixSize; i++) {
		CUDA_SAFE_CALL( hipMemcpy(&Aa[i * matrixSize], A[i], matrixSize*sizeof(float), hipMemcpyHostToDevice) );
		CUDA_SAFE_CALL( hipMemcpy(&Bb[i * matrixSize], B[i], matrixSize*sizeof(float), hipMemcpyHostToDevice) );
	}
	
	/*Timer*/
	timestamp t_start;
	
	/*Create Cuda grid*/
	dim3 dimBl(threads, threads);
	dim3 dimGr(FRACTION_CEILING(matrixSize, threads), FRACTION_CEILING(matrixSize, threads));
	
	/*Start the timer*/
	t_start = getTimestamp();
	
	
	/*Make the simulation*/
	for (i = 0; i < steps; i++) {
		if (i % 2 == 0) {
			/*From Aa to Bb*/
			heat<<<dimGr, dimBl>>>(Aa, Bb, matrixSize, matrixSize * matrixSize);
		}
		else {
			/*From Bb to Aa*/
			heat<<<dimGr, dimBl>>>(Bb, Aa, matrixSize, matrixSize * matrixSize);
		}
		/*synchronize threads*/
		CUDA_SAFE_CALL( hipDeviceSynchronize() );
	}
	
	/*Stop the timer*/
	float msecs = getElapsedtime(t_start);
	
	/*Copy data from device memory*/
	for (i = 0; i < matrixSize; i++) {
		CUDA_SAFE_CALL( hipMemcpy(A[i], &Aa[i * matrixSize], matrixSize*sizeof(float), hipMemcpyDeviceToHost) );
		CUDA_SAFE_CALL( hipMemcpy(B[i], &Bb[i * matrixSize], matrixSize*sizeof(float), hipMemcpyDeviceToHost) );
	}
	
	
	/*Clean up*/
	CUDA_SAFE_CALL( hipFree(Aa) );
	CUDA_SAFE_CALL( hipFree(Bb) );
	return msecs;
}